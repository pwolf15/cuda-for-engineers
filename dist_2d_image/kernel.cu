
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#define W 500
#define H 500
#define TX 32
#define TY 32

__device__ char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__global__
void distanceKernel(uchar4 *d_out, int w, int h, int2 pos)
{
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;
  const int i = r*w + c;
  if ((c >= w) || (r >= h)) return;

  const int d = sqrtf(((c-pos.x)*(c-pos.x))+((r-pos.y)*(r-pos.y)));
  const unsigned char intensity = clip(255 - d);

  d_out[i].x = intensity;
  d_out[i].y = intensity;
  d_out[i].z = intensity;
  d_out[i].w = 255;

  printf("%d\n", d_out[i].x);
}

int main()
{
  uchar4 *d_out = nullptr;
  uchar4 *out = (uchar4*)calloc(W*H, sizeof(uchar4));

  hipMalloc(&d_out, W*H*sizeof(uchar4));

  const int2 pos = {0,0};
  const dim3 blockSize{TX, TY};
  const int bx = (W + TX - 1) / TX;
  const int by = (H + TY - 1) / TY;
  const dim3 gridSize{bx, by};
  distanceKernel<<<gridSize, blockSize>>>(d_out, W, H, pos);

  hipMemcpy(out, d_out, W*H*sizeof(uchar4), hipMemcpyDeviceToHost);

  hipFree(d_out);
  free(out);
  return 0;
}