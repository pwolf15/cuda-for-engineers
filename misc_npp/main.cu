#include "hip/hip_runtime.h"
#define cimg_display 0
#include "CImg.h"
#include <hip/hip_runtime.h>
#include <npp.h>
#include <stdlib.h>
#define kNumCh 3
#include <iostream>

void sharpenNPP(Npp8u *arr, int w, int h)
{
  Npp8u *d_in = 0, *d_out = 0;
  Npp32f *d_filter = 0;
  const Npp32f filter[9] =
  {
    -1.0, -1.0, -1.0,
    -1.0,  9.0, -1.0,
    -1.0, -1.0, -1.0
  };
  hipMalloc(&d_out, kNumCh*w*h*sizeof(Npp8u));
  hipMalloc(&d_in, kNumCh*w*h*sizeof(Npp8u));
  hipMalloc(&d_filter, 9*sizeof(Npp32f));
  hipMemcpy(d_in, arr, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  hipMemcpy(d_filter, filter, 9*sizeof(Npp32f),
             hipMemcpyHostToDevice);
  const NppiSize oKernelSize = { 3, 3 };
  const NppiPoint oAnchor = {1, 1};
  const NppiSize oSrcSize = {w, h};
  const NppiPoint oSrcOffset = {0, 0};
  const NppiSize oSizeROI = {w, h};

  nppiFilterBorder32f_8u_C3R(d_in, kNumCh*w*sizeof(Npp8u), oSrcSize,
    oSrcOffset, d_out, kNumCh*w*sizeof(Npp8u), oSizeROI, d_filter,
    oKernelSize, oAnchor, NPP_BORDER_REPLICATE);
  
  hipMemcpy(arr, d_out, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_filter);
}

void permuteNPP(Npp8u *arr, int w, int h)
{
  Npp8u *d_in = 0, *d_out = 0;

  hipMalloc(&d_out, kNumCh*w*h*sizeof(Npp8u));
  hipMalloc(&d_in, kNumCh*w*h*sizeof(Npp8u));
  hipMemcpy(d_in, arr, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  const NppiSize oSizeROI = {w, h};
  const int aDstOrder[3] = {1,2,0};

  nppiSwapChannels_8u_C3R(d_in, kNumCh*w*sizeof(Npp8u), d_out,
    kNumCh*w*sizeof(Npp8u), oSizeROI, aDstOrder);
  
  hipMemcpy(arr, d_out, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

void sumNPP(Npp8u *arr, Npp8u* out, int w, int h)
{
  Npp8u *d_in = 0, *d_out = 0;

  hipMalloc(&d_out, kNumCh*w*h*sizeof(Npp8u));
  hipMalloc(&d_in, kNumCh*w*h*sizeof(Npp8u));
  hipMemcpy(d_in, arr, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  const NppiSize oSizeROI = {w, h};

  nppiAdd_8u_C3RSfs(d_in, kNumCh*w*sizeof(Npp8u), d_out,
    kNumCh*w*sizeof(Npp8u), d_out, kNumCh*w*sizeof(Npp8u), oSizeROI, 1);
  
  hipMemcpy(arr, d_out, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

void normNPP(Npp8u *arr, Npp8u* out, int w, int h)
{
  Npp8u *d_in = 0, *d_out = 0;

  hipMalloc(&d_out, kNumCh*w*h*sizeof(Npp8u));
  hipMalloc(&d_in, kNumCh*w*h*sizeof(Npp8u));
  hipMemcpy(d_in, arr, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  const NppiSize oSizeROI = {w, h};
  Npp64f *aNorm = 0;
  hipMalloc(&aNorm, 3*sizeof(Npp64f));
  Npp8u *pDeviceBuffer = 0;
  int bufferSize;
  nppiNormDiffL2GetBufferHostSize_8u_C3R(oSizeROI, &bufferSize);
  hipMalloc(&pDeviceBuffer, bufferSize);
  nppiNormDiff_L2_8u_C3R(d_in, kNumCh*w*sizeof(Npp8u), d_out,
    kNumCh*w*sizeof(Npp8u), oSizeROI, aNorm, pDeviceBuffer);
  Npp64f res[3];
  hipMemcpy(res, aNorm, 3*sizeof(Npp64f), hipMemcpyDeviceToHost);
  for (int i = 0; i < 3; ++i)
  {
    printf("%f\n", res[i]/(w*h));
  }

  float avg = 0, avg2 = 0;
  Npp8u minimum = 255;
  for (int i = 0; i < w*h*3; ++i)
  {
    avg += arr[i];
    avg2 += out[i];
    minimum = std::min(minimum, out[i]);
    if (arr[i] != out[i])
    {
      std::cout << "Different!" << std::endl;
      break;
    }
  }

  std::cout << "Average: " << avg / (w*h) << std::endl;
  std::cout << "Average 2: " << avg2 / (w*h) << std::endl;
  std::cout << "Minimum: " << minimum << std::endl;

  hipMemcpy(arr, d_out, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
}

void grayscaleNormNPP(Npp8u *arr, Npp8u* out, int w, int h)
{
  Npp8u *d_in = 0, *d_out = 0, *d_temp_gray1 = 0, *d_temp_gray2 = 0;

  hipMalloc(&d_out, kNumCh*w*h*sizeof(Npp8u));
  hipMalloc(&d_in, kNumCh*w*h*sizeof(Npp8u));
  hipMalloc(&d_temp_gray1, w*h*sizeof(Npp8u));
  hipMalloc(&d_temp_gray2, w*h*sizeof(Npp8u));

  hipMemcpy(d_in, arr, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  hipMemcpy(d_out, out, kNumCh*w*h*sizeof(Npp8u),
             hipMemcpyHostToDevice);
  const NppiSize oSizeROI = {w, h};
  Npp64f *pNormDiff = NULL;
  hipMalloc(&pNormDiff, sizeof(Npp64f));

  Npp8u* pDeviceBufferGray = NULL;
  int bufferSizeGray;
  nppiNormDiffL2GetBufferHostSize_8u_C1R(oSizeROI, &bufferSizeGray);
  hipMalloc(&pDeviceBufferGray, bufferSizeGray);

  // Convert to Grayscale
  nppiRGBToGray_8u_C3C1R(d_in, kNumCh*w*sizeof(Npp8u), d_temp_gray1,
    w*sizeof(Npp8u), oSizeROI);
  nppiRGBToGray_8u_C3C1R(d_out, kNumCh*w*sizeof(Npp8u), d_temp_gray2,
    w*sizeof(Npp8u), oSizeROI);

  // grayscale norm diff
  nppiNormDiff_L2_8u_C1R(d_temp_gray1, w*sizeof(Npp8u), d_temp_gray2,
    w*sizeof(Npp8u), oSizeROI, pNormDiff, pDeviceBufferGray);

  Npp8u *h_temp_gray1 = 0, *h_temp_gray2 = 0;

  h_temp_gray1 = (Npp8u*)malloc(w*h*sizeof(Npp8u));
  h_temp_gray2 = (Npp8u*)malloc(w*h*sizeof(Npp8u));
  hipMemcpy(h_temp_gray1, d_temp_gray1, w*h*sizeof(Npp8u),
             hipMemcpyDeviceToHost);
  hipMemcpy(h_temp_gray2, d_temp_gray2, w*h*sizeof(Npp8u),
             hipMemcpyDeviceToHost);

  float avg = 0, avg2 = 0;
  Npp8u minimum = 255;
  for (int i = 0; i < w*h; ++i)
  {
    avg += arr[i];
    avg2 += out[i];
    minimum = std::min(minimum, out[i]);
    if (arr[i] != out[i])
    {
      std::cout << "Different!" << std::endl;
      break;
    }
  }

  std::cout << "Average: " << avg / (w*h) << std::endl;
  std::cout << "Average 2: " << avg2 / (w*h) << std::endl;
  std::cout << "Minimum: " << minimum << std::endl;
  
  Npp64f res = 0;
  hipMemcpy(&res, pNormDiff, sizeof(Npp64f), hipMemcpyDeviceToHost);
  printf("%f\n", res);
}

int main()
{
  cimg_library::CImg<unsigned char> image("/home/pwolf/dev/cuda_for_engineers/misc_npp/build/Tricoloring.png");
  const int w = image.width();
  const int h = image.height();
  Npp8u *arr = (Npp8u*)malloc(kNumCh*w*h*sizeof(Npp8u));

  for (int r = 0; r < h; ++r)
  {
    for (int c = 0; c < w; ++c)
    {
      for (int ch = 0; ch < kNumCh; ++ch)
      {
        arr[kNumCh*(r*w + c) + ch] = image(c, r, ch);
      }
    }
  }

  // create copy of original
  Npp8u *orig = (Npp8u*)malloc(kNumCh*w*h*sizeof(Npp8u));
  memcpy((void*)orig, (void*)arr, kNumCh*w*h*sizeof(Npp8u));

  // sharpened
  sharpenNPP(arr, w, h);

  // create copy of sharpened
  Npp8u *sharpened = (Npp8u*)malloc(kNumCh*w*h*sizeof(Npp8u));
  memcpy((void*)sharpened, (void*)arr, kNumCh*w*h*sizeof(Npp8u));

  for (int r = 0; r < h; ++r)
  {
    for (int c = 0; c < w; ++c)
    {
      for (int ch = 0; ch < kNumCh; ++ch)
      {
        image(c, r, ch) = arr[kNumCh*(r*w + c) + ch];
      }
    }
  }

  image.save_bmp("out.bmp");

  // // permuted
  // permuteNPP(arr, w, h);

  // for (int r = 0; r < h; ++r)
  // {
  //   for (int c = 0; c < w; ++c)
  //   {
  //     for (int ch = 0; ch < kNumCh; ++ch)
  //     {
  //       image(c, r, ch) = arr[kNumCh*(r*w + c) + ch];
  //     }
  //   }
  // }

  // // create copy of permuted
  // Npp8u *permuted = (Npp8u*)malloc(kNumCh*w*h*sizeof(Npp8u));
  // memcpy((void*)permuted, (void*)arr, kNumCh*w*h*sizeof(Npp8u));

  // // sum original and permuted
  // sumNPP(arr, orig, w, h);

  // for (int r = 0; r < h; ++r)
  // {
  //   for (int c = 0; c < w; ++c)
  //   {
  //     for (int ch = 0; ch < kNumCh; ++ch)
  //     {
  //       image(c, r, ch) = arr[kNumCh*(r*w + c) + ch];
  //     }
  //   }
  // }

  // image.save_bmp("sum.bmp");

  // std::cout << "Norm with itself" << std::endl;
  // normNPP(orig, orig, w, h);      // compare with itself

  std::cout << "Norm with sharpened" << std::endl;
  normNPP(sharpened, orig, w, h); // compare wih sharpened

  // std::cout << "Norm with color-swapped" << std::endl;
  // normNPP(permuted, orig, w, h); // compare with color-swapped
  // std::cout << "Grayscale: norm with color-swapped" << std::endl;
  grayscaleNormNPP(sharpened, orig, w, h); // compare with grayscale color-swapped

  free(arr);
  return 0;
}
